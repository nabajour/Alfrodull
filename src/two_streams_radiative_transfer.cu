#include "hip/hip_runtime.h"
// ==============================================================================
// This file is part of THOR.
//
//     THOR is free software : you can redistribute it and / or modify
//     it under the terms of the GNU General Public License as published by
//     the Free Software Foundation, either version 3 of the License, or
//     (at your option) any later version.
//
//     THOR is distributed in the hope that it will be useful,
//     but WITHOUT ANY WARRANTY; without even the implied warranty of
//     MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the
//     GNU General Public License for more details.
//
//     You find a copy of the GNU General Public License in the main
//     THOR directory under <license.txt>.If not, see
//     <http://www.gnu.org/licenses/>.
// ==============================================================================
//
// Two stream radiative transfer
//
//
//
// Method: Helios Two Stream algorithm
//
//
// Known limitations: - Runs in a single GPU.
//
// Known issues: None
//
//
// If you use this code please cite the following reference:
//
//       [1] Mendonca, J.M., Grimm, S.L., Grosheintz, L., & Heng, K., ApJ, 829, 115, 2016
//
// Current Code Owner: Joao Mendonca, EEG. joao.mendonca@csh.unibe.ch
//
// History:
// Version Date       Comment
// ======= ====       =======
//
//
//
////////////////////////////////////////////////////////////////////////


#include "two_streams_radiative_transfer.h"

#include "binary_test.h"
#include "debug.h"
#include "debug_helpers.h"

#include "alfrodull_engine.h"

#include "physics_constants.h"

#include "directories.h"
#include "storage.h"

#include <string>

#include <functional>
#include <map>

#include "insolation.h"

#include "math_helpers.h"

USE_BENCHMARK();


using std::string;


// show progress bar
#define COLUMN_LOOP_PROGRESS_BAR

// debugging printout
#define DEBUG_PRINTOUT_ARRAYS
// dump TP profile to run in HELIOS for profile comparison
#define DUMP_HELIOS_TP
// stride for column TP profile dump
#ifdef DUMP_HELIOS_TP
const int HELIOS_TP_STRIDE = 1;
#endif // DUMP_HELIOS_TP


//***************************************************************************************************

const char PBSTR[] = "||||||||||||||||||||||||||||||||||||||||||||||||||||||||||||";
const int  PBWIDTH = 60;

void print_progress(double percentage) {
    int val  = (int)(percentage * 100);
    int lpad = (int)(percentage * PBWIDTH);
    int rpad = PBWIDTH - lpad;
    printf("\r%3d%% [%.*s%*s]", val, lpad, PBSTR, rpad, "");
    fflush(stdout);
}

two_streams_radiative_transfer::two_streams_radiative_transfer() {
}

two_streams_radiative_transfer::~two_streams_radiative_transfer() {
}

void two_streams_radiative_transfer::print_config() {
    log::printf("    Tstar: %g\n", T_star);
    log::printf("    T_intern: %g", T_internal);
    log::printf("    Alf_iso: %s\n", iso ? "true" : "false");
    log::printf("    Alf_real_star: %s\n", real_star ? "true" : "false");
    log::printf("    Alf_fake_opac: %f\n", fake_opac);

    log::printf("    Alf_stellar_spectrum: %s\n", stellar_spectrum_file.c_str());

    log::printf("    Alf_thomas: %s\n", thomas ? "true" : "false");
    log::printf("    Alf_scat_single_walk: %s\n", scat_single_walk ? "true" : "false");
    log::printf("    Alf_exp_opac_offset: %g\n", experimental_opacities_offset);

    log::printf("    Alf_g_0: %f\n", g_0);
    //    log::printf("    Alf_diffusivity: %f\n", diffusivity);
    log::printf("    Alf_epsilon_2: %f\n", epsilon_2);

    log::printf("    Alf_G_pm_max_limiter: %s\n", G_pm_limiter ? "true" : "false");
    log::printf("    Alf_G_pm_denom_limit: %f\n", G_pm_denom_limit);
    log::printf("    Alf_G_pm_mu_star_increment: %f\n", mu_star_wiggle_increment);
    log::printf("    Alf_direct_beam_angle_limit: %f\n", mu_star_limit_degrees);

    log::printf("    Alf_scat: %s\n", scat ? "true" : "false");
    log::printf("    Alf_scat_corr: %s\n", scat_corr ? "true" : "false");
    log::printf("    R_star: %f [R_SUN]\n", R_star_config);
    log::printf("    planet star dist: %f [au]\n", planet_star_dist_config);

    log::printf("    Alf_dir_beam: %s\n", dir_beam ? "true" : "false");
    log::printf("    Alf_geom_zenith_corr: %s\n", geom_zenith_corr ? "true" : "false");

    log::printf("    Alf_w_0_limit: %f\n", w_0_limit);
    log::printf("    Alf_i2s_transition: %f\n", i2s_transition);
    log::printf("    Alf_opacities_file: %s\n", opacities_file.c_str());
    log::printf("    Alf_compute_every_nstep: %d\n", compute_every_n_iteration);

    log::printf("    Alf_clouds: %s\n", clouds ? "true" : "false");
    log::printf("    Alf_fcloud: %g\n", fcloud);
    log::printf("    Alf_cloudfile: %s\n", cloud_filename.c_str());
    log::printf("    Alf_store_w0_g0: %s\n", store_w0_g0 ? "true" : "false");
    // spinup-spindown parameters
    log::printf("    Spin up start step          = %d.\n", spinup_start_step);
    log::printf("    Spin up stop step           = %d.\n", spinup_stop_step);
    log::printf("    Spin down start step        = %d.\n", spindown_start_step);
    log::printf("    Spin down stop step         = %d.\n", spindown_stop_step);
}

bool two_streams_radiative_transfer::configure(config_file& config_reader) {
    // variables reused from DG
    config_reader.append_config_var("Tstar", T_star, T_star);
    config_reader.append_config_var("Tint", T_internal, T_internal);
    config_reader.append_config_var(
        "planet_star_dist", planet_star_dist_config, planet_star_dist_config);
    config_reader.append_config_var("radius_star", R_star_config, R_star_config);

    config_reader.append_config_var("Alf_thomas", thomas, thomas);
    config_reader.append_config_var("Alf_scat_single_walk", scat_single_walk, scat_single_walk);
    config_reader.append_config_var(
        "Alf_exp_opac_offset", experimental_opacities_offset, experimental_opacities_offset);
    config_reader.append_config_var("Alf_iso", iso, iso);
    config_reader.append_config_var("Alf_real_star", real_star, real_star);
    config_reader.append_config_var(
        "Alf_stellar_spectrum", stellar_spectrum_file, stellar_spectrum_file);
    config_reader.append_config_var("Alf_fake_opac", fake_opac, fake_opac);

    config_reader.append_config_var("Alf_g_0", g_0, g_0);
    config_reader.append_config_var("Alf_diffusivity", diffusivity, diffusivity);
    config_reader.append_config_var("Alf_G_pm_max_limiter", G_pm_limiter, G_pm_limiter);
    config_reader.append_config_var("Alf_G_pm_denom_limit", G_pm_denom_limit, G_pm_denom_limit);
    config_reader.append_config_var(
        "Alf_G_pm_mu_star_increment", mu_star_wiggle_increment, mu_star_wiggle_increment);
    config_reader.append_config_var(
        "Alf_direct_beam_angle_limit", mu_star_limit_degrees, mu_star_limit_degrees);
    config_reader.append_config_var("Alf_scat", scat, scat);
    config_reader.append_config_var("Alf_scat_corr", scat_corr, scat_corr);

    config_reader.append_config_var("Alf_dir_beam", dir_beam, dir_beam);
    config_reader.append_config_var("Alf_geom_zenith_corr", geom_zenith_corr, geom_zenith_corr);
    config_reader.append_config_var("Alf_i2s_transition", i2s_transition, i2s_transition);

    config_reader.append_config_var("Alf_opacities_file", opacities_file, opacities_file);
    config_reader.append_config_var(
        "Alf_compute_every_nstep", compute_every_n_iteration, compute_every_n_iteration);

    // spin up spin down
    config_reader.append_config_var("Alf_spinup_start", spinup_start_step, spinup_start_step);
    config_reader.append_config_var("Alf_spinup_stop", spinup_stop_step, spinup_stop_step);
    config_reader.append_config_var("Alf_spindown_start", spindown_start_step, spindown_start_step);
    config_reader.append_config_var("Alf_spindown_stop", spindown_stop_step, spindown_stop_step);

    config_reader.append_config_var("Alf_clouds", clouds, clouds);
    config_reader.append_config_var("Alf_fcloud", fcloud, fcloud);
    config_reader.append_config_var("Alf_cloudfile", cloud_filename, cloud_filename);

    config_reader.append_config_var("Alf_store_w0_g0", store_w0_g0, store_w0_g0);

    return true;
}


bool two_streams_radiative_transfer::initialise_memory(
    const ESP&               esp,
    device_RK_array_manager& phy_modules_core_arrays) {
    bool out = true;
    nlayer   = esp.nv; // (??) TODO: check

    // TODO: understand what needs to be stored per column. and what can be global for internal conputation
    // what needs to be passed outside or stored should be global, others can be per column

    R_star_SI = R_star_config * R_SUN;

    planet_star_dist_SI = planet_star_dist_config * AU;


    // as set in host_functions.set_up_numerical_parameters
    // w_0_limit
    w_0_limit = 1.0 - 1e-14;

    double f_factor = 1.0;

    epsi = 1.0 / diffusivity;

    alf.thomas = thomas;

    alf.G_pm_limiter             = G_pm_limiter;
    alf.G_pm_denom_limit         = G_pm_denom_limit;
    alf.mu_star_wiggle_increment = mu_star_wiggle_increment;

    double mu_star_limit = cos((90.0 + mu_star_limit_degrees) / 180.0 * M_PI);

    alf.set_parameters(nlayer,              // const int&    nlayer_,
                       iso,                 // const bool&   iso_,
                       T_star,              // const double& T_star_,
                       real_star,           // const bool&   real_star_,
                       fake_opac,           // const double& fake_opac_,
                       g_0,                 // const double& g_0_,
                       epsi,                // const double& epsi_,
                       epsilon_2,           // const double& epsilon_2_,
                       scat,                // const bool&   scat_,
                       scat_corr,           // const bool&   scat_corr_,
                       0.0,                 // const double& R_planet_, filled in later
                       R_star_SI,           // const double& R_star_,
                       planet_star_dist_SI, // const double& a_,
                       dir_beam,            // const bool&   dir_beam_,
                       geom_zenith_corr,    // const bool&   geom_zenith_corr_,
                       f_factor,            // const double& f_factor_,
                       w_0_limit,           // const double& w_0_limit_,
                       i2s_transition,      // const double& i2s_transition_,
                       mu_star_limit,
                       false); // const bool&   debug_

    // initialise opacities table -> gives frequency bins
    // set opacity offset for test
    alf.set_experimental_opacity_offset(experimental_opacities_offset);

    alf.load_opacities(opacities_file);

    hipDeviceSynchronize();
    log::printf("Loaded opacities, using %d bins with %d weights per bin\n",
                alf.opacities.nbin,
                alf.opacities.ny);

    alf.allocate_internal_variables();

    int ninterface         = nlayer + 1;
    int nlayer_plus1       = nlayer + 1;
    int nbin               = alf.opacities.nbin;
    int ny                 = alf.opacities.ny;
    int nlayer_nbin        = nlayer * nbin;
    int ninterface_nbin    = ninterface * nbin;
    int ninterface_wg_nbin = ninterface * ny * nbin;

    if (real_star) {
        // load star flux.
        std::printf("Using Stellar Flux file %s\n", stellar_spectrum_file.c_str());
        star_flux.allocate(nbin);
        if (!path_exists(stellar_spectrum_file)) {
            log::printf("Stellar spectrum file not found: %s\n", stellar_spectrum_file.c_str());
            exit(EXIT_FAILURE);
        }

        double lambda_spectrum_scale = 1e-2;
        double flux_scale            = 1e-1;

        storage s(stellar_spectrum_file, true);
        if (s.has_table("wavelength") && s.has_table("flux")) {
            std::unique_ptr<double[]> lambda_ptr  = nullptr;
            int                       lambda_size = 0;

            std::unique_ptr<double[]> flux_ptr  = nullptr;
            int                       flux_size = 0;

            s.read_table("wavelength", lambda_ptr, lambda_size);
            s.read_table("flux", flux_ptr, flux_size);

            if (lambda_size != nbin || lambda_size != flux_size) {
                log::printf("Wrong size for stellar size arrays\n");
                log::printf("Lambda: %d\n", lambda_size);
                log::printf("Flux: %d\n", flux_size);
                log::printf("nbin: %d\n", nbin);
                exit(EXIT_FAILURE);
            }

            bool                      lambda_check = true;
            double                    epsilon      = 1e-4;
            std::shared_ptr<double[]> star_flux_h  = star_flux.get_host_data_ptr();
            for (int i = 0; i < nbin; i++) {
                star_flux_h[i] = flux_ptr[i] * flux_scale;
                bool check =
                    fabs(lambda_ptr[i] * lambda_spectrum_scale - alf.opacities.data_opac_wave[i])
                        / alf.opacities.data_opac_wave[i]
                    < epsilon;

                if (!check)
                    printf("Missmatch in wavelength at idx [%d] l_spectrum(%g) != l_opac(%g) \n",
                           i,
                           lambda_ptr[i] * lambda_spectrum_scale,
                           alf.opacities.data_opac_wave[i]);
                lambda_check &= check;
            }

            star_flux.put();

            if (!lambda_check) {
                log::printf("wavelength points mismatch between stellar spectrum and opacities\n");
                exit(EXIT_FAILURE);
            }
        }
        else {
            log::printf("table wavelength or flux not found in stellar flux file\n");
            exit(EXIT_FAILURE);
        }
        printf("Stellar flux loaded\n");
    }

    // TODO: allocate here. Should be read in in case of real_star == true
    //    star_flux.allocate(nbin);
    // allocate interface state variables to be interpolated


    pressure_int.allocate(ninterface);
    temperature_int.allocate(ninterface);
    temperature_lay.allocate(nlayer_plus1);


    F_down_wg.allocate(ninterface_wg_nbin);
    F_up_wg.allocate(ninterface_wg_nbin);
    F_dir_wg.allocate(ninterface_wg_nbin);

    if (!iso) {
        Fc_down_wg.allocate(ninterface_wg_nbin);
        Fc_up_wg.allocate(ninterface_wg_nbin);
        Fc_dir_wg.allocate(ninterface_wg_nbin);
    }

    F_down_tot.allocate(esp.point_num * ninterface);
    F_up_tot.allocate(esp.point_num * ninterface);
    F_dir_tot.allocate(esp.point_num * ninterface);
    F_down_band.allocate(ninterface_nbin);
    F_up_band.allocate(ninterface_nbin);
    F_dir_band.allocate(ninterface_nbin);
    // TODO: check, ninterface or nlayers ?
    F_net.allocate(esp.point_num * ninterface);

    F_up_TOA_spectrum.allocate(esp.point_num * nbin);


    Qheat.allocate(esp.point_num * nlayer);

    if (store_w0_g0) {
        // output for storage
        g0_tot.allocate(esp.point_num * nlayer_nbin);
        w0_tot.allocate(esp.point_num * nlayer_nbin);
        // output for storage
        g0_tot.zero();
        w0_tot.zero();
    }

    if (clouds) {
        // load cloud file
        alf.cloud_opacities.load(cloud_filename);

        alf.set_clouds_data(clouds,
                            *alf.cloud_opacities.dev_abs_cross_sections,
                            *alf.cloud_opacities.dev_abs_cross_sections,
                            *alf.cloud_opacities.dev_scat_cross_sections,
                            *alf.cloud_opacities.dev_scat_cross_sections,
                            *alf.cloud_opacities.dev_asymmetry,
                            *alf.cloud_opacities.dev_asymmetry,
                            fcloud);
    }
    else {
        // all clouds set to zero. Not used.
        g_0_tot_lay.allocate(nbin);
        g_0_tot_int.allocate(nbin);
        cloud_abs_cross_lay.allocate(nbin);
        cloud_abs_cross_int.allocate(nbin);
        cloud_scat_cross_lay.allocate(nbin);
        cloud_scat_cross_int.allocate(nbin);


        g_0_tot_lay.zero();
        g_0_tot_int.zero();
        cloud_abs_cross_lay.zero();
        cloud_abs_cross_int.zero();
        cloud_scat_cross_lay.zero();
        cloud_scat_cross_int.zero();

        alf.set_clouds_data(clouds,
                            *cloud_abs_cross_lay,
                            *cloud_abs_cross_int,
                            *cloud_scat_cross_lay,
                            *cloud_scat_cross_int,
                            *g_0_tot_lay,
                            *g_0_tot_int,
                            fcloud);
    }


    hipError_t err = hipGetLastError();

    // Check device query
    if (err != hipSuccess) {
        log::printf("[%s:%d] CUDA error check reports error: %s\n",
                    __FILE__,
                    __LINE__,
                    hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

#ifdef BENCHMARKING
    std::map<string, output_def> debug_arrays = {
        {"F_net", {F_net.ptr_ref(), esp.point_num * ninterface, "Fnet", "Fn", true, dummy}},

        {"F_up_tot",
         {F_up_tot.ptr_ref(), esp.point_num * ninterface, "Fuptot", "Fut", true, dummy}},
        {"F_down_tot",
         {F_down_tot.ptr_ref(), esp.point_num * ninterface, "Fdowntot", "Fdt", true, dummy}},
        {"F_up_wg", {F_up_wg.ptr_ref(), ninterface_wg_nbin, "Fupwg", "Fuw", true, dummy}},
        {"F_down_wg", {F_down_wg.ptr_ref(), ninterface_wg_nbin, "Fdownwg", "Fdw", true, dummy}},
        {"F_up_band", {F_up_band.ptr_ref(), ninterface_nbin, "Fupband", "Fub", true, dummy}},
        {"F_down_band", {F_down_band.ptr_ref(), ninterface_nbin, "Fdownband", "Fdb", true, dummy}},
        {"F_dir_wg", {F_dir_wg.ptr_ref(), ninterface_wg_nbin, "Fdirwg", "Fdirw", true, dummy}},

        {"F_dir_band", {F_dir_band.ptr_ref(), ninterface_nbin, "Fdirband", "Fdib", true, dummy}},


        {"T_lay", {temperature_lay.ptr_ref(), nlayer_plus1, "T_lay", "Tl", true, dummy}},
        {"T_int", {temperature_int.ptr_ref(), ninterface, "T_int", "Ti", true, dummy}},
        {"P_int", {pressure_int.ptr_ref(), ninterface, "P_int", "Pi", true, dummy}},

        //        {"col_mu_star", {col_mu_star.ptr_ref(), esp.point_num, "col_mu_star", "cMu", true, dummy}},
        {"AlfQheat", {Qheat.ptr_ref(), esp.point_num * nlayer, "AlfQheat", "aQh", true, dummy}}};
    BENCH_POINT_REGISTER_PHY_VARS(debug_arrays, (), ());
#endif // BENCHMARKING
    return out;
}

bool two_streams_radiative_transfer::initial_conditions(const ESP&             esp,
                                                        const SimulationSetup& sim,
                                                        storage*               s) {
    if (spinup_start_step > -1 || spinup_stop_step > -1) {
        if (spinup_stop_step < spinup_start_step)
            printf("Alf: inconsistent spinup_start (%d) and spinup_stop (%d) values\n",
                   spinup_start_step,
                   spinup_stop_step);
    }
    if (spindown_start_step > -1 || spindown_stop_step > -1) {
        if (spindown_stop_step < spindown_start_step)
            printf("Alf: inconsistent spindown_start (%d) and spindown_stop (%d) values\n",
                   spindown_start_step,
                   spindown_stop_step);
    }

    bool out = true;
    // what should be initialised here and what is to initialise at each loop ?
    // what to initialise here and what to do in initialise memory ?

    // this is only known here, comes from sim setup.
    alf.R_planet = sim.A;
    cuda_check_status_or_exit(__FILE__, __LINE__);
    // initialise planck tables
    alf.prepare_planck_table();
    log::printf("Built Planck Table for %d bins, Star temp %g K\n", alf.opacities.nbin, alf.T_star);
    // initialise alf

    // TODO: where to do this, check
    // TODO where does starflux come from?
    // correct_incident_energy

    alf.correct_incident_energy(*star_flux, real_star, true);

    // internal flux from internal temperature
    F_intern = STEFANBOLTZMANN * pow(T_internal, 4);

    cuda_check_status_or_exit(__FILE__, __LINE__);

    // request insolation computation
    esp.insolation.set_require();
    return out;
}

// initialise delta_colmass arrays from pressure
// same as helios.source.host_functions.construct_grid
__global__ void initialise_delta_colmass_noniso(double* delta_col_mass_upper,
                                                double* delta_col_mass_lower,
                                                double* pressure_lay,
                                                double* pressure_int,
                                                double  gravit,
                                                int     num_layers) {
    int layer_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (layer_idx < num_layers) {
        delta_col_mass_upper[layer_idx] =
            (pressure_lay[layer_idx] - pressure_int[layer_idx + 1]) / gravit;
        delta_col_mass_lower[layer_idx] =
            (pressure_int[layer_idx] - pressure_lay[layer_idx]) / gravit;
    }
}

// initialise delta_colmass arrays from pressure
// same as helios.source.host_functions.construct_grid
__global__ void initialise_delta_colmass_iso(double* delta_col_mass,
                                             double* pressure_int,
                                             double  gravit,
                                             int     num_layers) {
    int layer_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (layer_idx < num_layers) {
        delta_col_mass[layer_idx] =
            (pressure_int[layer_idx] - pressure_int[layer_idx + 1]) / gravit;
    }
}


// single column pressure and temperature interpolation from layers to interfaces
// needs to loop from 0 to number of interfaces (nvi = nv+1)
// same as profX_RT
__global__ void interpolate_temperature_and_pressure(double* temperature_lay,      // out
                                                     double* temperature_lay_thor, // in
                                                     double* temperature_int,      // out
                                                     double* pressure_lay,         // in
                                                     double* pressure_int,         // out
                                                     double* density,              // in
                                                     double* altitude_lay,         // in
                                                     double* altitude_int,         // in
                                                     double  T_intern,
                                                     double  gravit,
                                                     int     num_layers) {
    int int_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Prepare temperature array with T_intern
    // TODO: check this, HELIOS uses temperature_lay[nlayer] as BOA value (also indexed as temperture_lay[numinterfaces - 1])
    // try helios style
    // printf("-intidx: %d/%d\n", int_idx, num_layers);
    if (int_idx < num_layers) {
        // printf("intidx: %d/%d\n", int_idx, num_layers);
        temperature_lay[int_idx] = temperature_lay_thor[int_idx];
    }
    else if (int_idx == num_layers) {
        //printf("intidx: %d/%d %g *\n", int_idx, num_layers, T_intern);
        temperature_lay[num_layers] = T_intern;
    }

    // compute interface values
    if (int_idx == 0) {
        // extrapolate to lower boundary
        double psm =
            pressure_lay[1]
            - density[0] * gravit * (2 * altitude_int[0] - altitude_lay[0] - altitude_lay[1]);

        double ps = 0.5 * (pressure_lay[0] + psm);

        pressure_int[0]    = ps;
        temperature_int[0] = T_intern;
    }
    else if (int_idx == num_layers) {
        // extrapolate to top boundary
        double pp = pressure_lay[num_layers - 2]
                    + (pressure_lay[num_layers - 1] - pressure_lay[num_layers - 2])
                          / (altitude_lay[num_layers - 1] - altitude_lay[num_layers - 2])
                          * (2 * altitude_int[num_layers] - altitude_lay[num_layers - 1]
                             - altitude_lay[num_layers - 2]);
        if (pp < 0.0)
            pp = 0.0; //prevents pressure at the top from becoming negative
        double ptop = 0.5 * (pressure_lay[num_layers - 1] + pp);

        pressure_int[num_layers] = ptop;
        // extrapolate to top interface
        temperature_int[num_layers] =
            temperature_lay_thor[num_layers - 1]
            + 0.5 * (temperature_lay_thor[num_layers - 1] - temperature_lay_thor[num_layers - 2]);
    }
    else if (int_idx < num_layers) {
        // interpolation between layers
        // Helios computes gy taking the middle between the layers. We can have non uniform Z levels,
        // so linear interpolation
        double xi       = altitude_int[int_idx];
        double xi_minus = altitude_lay[int_idx - 1];
        double xi_plus  = altitude_lay[int_idx];
        double a        = (xi - xi_plus) / (xi_minus - xi_plus);
        double b        = (xi - xi_minus) / (xi_plus - xi_minus);

        pressure_int[int_idx] = pressure_lay[int_idx - 1] * a + pressure_lay[int_idx] * b;

        temperature_int[int_idx] =
            temperature_lay_thor[int_idx - 1] * a + temperature_lay_thor[int_idx] * b;
    }
}

__global__ void
increment_Qheat(double* Qheat_global, double* Qheat, double scaling, int num_sample) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_sample) {
        // delta_flux/delta_z
        Qheat_global[idx] += scaling * Qheat[idx];
    }
}

__global__ void compute_column_Qheat(double* F_net, // net flux, layer
                                     double* z_int,
                                     double* Qheat,
                                     double  F_intern,
                                     int     num_layers) {
    int layer_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (layer_idx == 0) {
        // delta_flux/delta_z
        // F_net positive in upward direction (F_up - F_down)
        // F_intern positive, flux out of bottom surface
        // Qheat negative when net flux differential out of layer is positive
        Qheat[layer_idx] = -((F_net[1] - (F_net[0] + F_intern))) / (z_int[1] - z_int[0]);
    }
    else if (layer_idx < num_layers) {
        // delta_flux/delta_z
        Qheat[layer_idx] =
            -(F_net[layer_idx + 1] - F_net[layer_idx]) / (z_int[layer_idx + 1] - z_int[layer_idx]);
    }
}


bool two_streams_radiative_transfer::phy_loop(ESP&                   esp,
                                              const SimulationSetup& sim,
                                              int                    nstep, // Step number
                                              double                 time_step)             // Time-step [s]
{
    bool run      = true;
    qheat_scaling = 1.0;

    if (spinup_start_step > -1 && spinup_stop_step > -1) {
        if (nstep < spinup_start_step) // before spinup
        {
            run           = false;
            qheat_scaling = 0.0;
        }
        else if ((nstep >= spinup_start_step) && (nstep <= spinup_stop_step)) // during spinup
        {
            double x = (double)(nstep - spinup_start_step)
                       / (double)(spinup_stop_step - spinup_start_step);
            qheat_scaling = (1 + sin(M_PI * x - M_PI / 2.0)) / 2.0;
            run           = true;
        }
    }

    if (spindown_start_step > -1 && spindown_stop_step > -1) {
        if ((nstep >= spindown_start_step) && (nstep <= spindown_stop_step)) {
            double x = (double)(nstep - spindown_start_step)
                       / (double)(spindown_stop_step - spindown_start_step);
            qheat_scaling = 1.0 - (1 + sin(M_PI * x - M_PI / 2.0)) / 2.0;
            run           = true;
        }
        else if (nstep >= spindown_stop_step) {
            run           = false;
            qheat_scaling = 0.0;
        }
    }

    if (run) {

        alf.debug_nstep = nstep;

        const int num_blocks = 256;

        if (nstep % compute_every_n_iteration == 0 || start_up) {
            std::shared_ptr<double[]> col_cos_zenith_angle_h =
                esp.insolation.get_host_cos_zenith_angles();

            Qheat.zero();
            F_down_tot.zero();
            F_up_tot.zero();
            F_dir_tot.zero();
            F_up_band.zero();
            F_dir_band.zero();
            F_net.zero();

            g0_tot.zero();
            w0_tot.zero();

            printf("\r\n");
            printf("\r\n");
            printf("\r\n");
            hipDeviceSynchronize();
            cuda_check_status_or_exit(__FILE__, __LINE__);
            int nbin = alf.opacities.nbin;
            // loop on columns
            for (int column_idx = 0; column_idx < esp.point_num; column_idx++) {
                alf.debug_col_idx = column_idx;
                hipDeviceSynchronize();
                cuda_check_status_or_exit(__FILE__, __LINE__);
#ifdef COLUMN_LOOP_PROGRESS_BAR
                print_progress((column_idx + 1.0) / double(esp.point_num));
#endif // COLUMN_LOOP_PROGRESS_BAR

                F_up_wg.zero();
                F_down_wg.zero();
                F_dir_wg.zero();
                if (iso) {
                }
                else {
                    Fc_down_wg.zero();
                    Fc_up_wg.zero();

                    Fc_dir_wg.zero();
                }


                alf.reset();

                pressure_int.zero();
                temperature_int.zero();
                temperature_lay.zero();

                // g_0_tot_lay.zero();
                // g_0_tot_int.zero();
                // cloud_abs_cross_lay.zero();
                // cloud_abs_cross_int.zero();
                // cloud_scat_cross_lay.zero();
                // cloud_scat_cross_int.zero();
                int num_layers = esp.nv;


                // TODO: get column offset
                int column_offset = column_idx * num_layers;


                double gravit = sim.Gravit;
                // fetch column values

                // TODO: check that I got the correct ones between slow and fast modes
                double* column_layer_temperature_thor = &(esp.temperature_d[column_offset]);
                double* column_layer_pressure         = &(esp.pressure_d[column_offset]);
                double* column_density                = &(esp.Rho_d[column_offset]);
                // initialise interpolated T and P
                hipDeviceSynchronize();
                cuda_check_status_or_exit(__FILE__, __LINE__);
                // use mu_star per column
                double mu_star = -col_cos_zenith_angle_h[column_idx];
#ifdef DUMP_HELIOS_TP
                hipDeviceSynchronize();
                cuda_check_status_or_exit(__FILE__, __LINE__);

                // dump a TP profile for HELIOS input
                if (column_idx % HELIOS_TP_STRIDE == 0) {
                    std::string DBG_OUTPUT_DIR = esp.get_output_dir()
                                                 + "/alfprof/"
                                                   "step_"
                                                 + std::to_string(nstep) + "/column_"
                                                 + std::to_string(column_idx) + "/";
                    create_output_dir(DBG_OUTPUT_DIR);

                    double                    lon = esp.lonlat_h[column_idx * 2 + 0] * 180 / M_PI;
                    double                    lat = esp.lonlat_h[column_idx * 2 + 1] * 180 / M_PI;
                    std::shared_ptr<double[]> pressure_h =
                        get_cuda_data(column_layer_pressure, esp.nv);
                    std::shared_ptr<double[]> temperature_h =
                        get_cuda_data(column_layer_temperature_thor, esp.nv);


                    double p_toa = pressure_h[esp.nv - 1];
                    double p_boa = pressure_h[0];


                    // Print out initial TP profile
                    string output_file_name = DBG_OUTPUT_DIR + "tpprofile_init.dat";

                    FILE*  tp_output_file = fopen(output_file_name.c_str(), "w");
                    string comment = "# Helios TP profile table at lat: [" + std::to_string(lon)
                                     + "] lon: [" + std::to_string(lat) + "] mustar: ["
                                     + std::to_string(mu_star) + "] P_BOA: ["
                                     + std::to_string(p_boa) + "] P_TOA: [" + std::to_string(p_toa)
                                     + "]\n";

                    fprintf(tp_output_file, comment.c_str());
                    fprintf(tp_output_file, "#\tT[K]\tP[bar]\n");

                    for (int i = 0; i < esp.nv; i++) {
                        fprintf(tp_output_file,
                                "%#.6g\t%#.6g\n",
                                temperature_h[i],
                                pressure_h[i] / 1e5);
                    }

                    fclose(tp_output_file);
                }
#endif // DUMP_HELIOS_TP
                hipDeviceSynchronize();
                cuda_check_status_or_exit(__FILE__, __LINE__);
                interpolate_temperature_and_pressure<<<((num_layers + 1) / num_blocks) + 1,
                                                       num_blocks>>>(*temperature_lay,
                                                                     column_layer_temperature_thor,
                                                                     *temperature_int,
                                                                     column_layer_pressure,
                                                                     *pressure_int,
                                                                     column_density,
                                                                     esp.Altitude_d,
                                                                     esp.Altitudeh_d,
                                                                     T_internal,
                                                                     gravit,
                                                                     num_layers);
                hipDeviceSynchronize();
                cuda_check_status_or_exit(__FILE__, __LINE__);

                BENCH_POINT_I_S(
                    nstep, column_idx, "Alf_interpTnP", (), ("T_lay", "T_int", "P_int"));

#ifdef DUMP_HELIOS_TP
                // dump a TP profile for HELIOS input
                if (column_idx % HELIOS_TP_STRIDE == 0) {
                    std::string DBG_OUTPUT_DIR = esp.get_output_dir()
                                                 + "/alfprof/"
                                                   "step_"
                                                 + std::to_string(nstep) + "/column_"
                                                 + std::to_string(column_idx) + "/";
                    create_output_dir(DBG_OUTPUT_DIR);

                    double lon = esp.lonlat_h[column_idx * 2 + 0] * 180 / M_PI;
                    double lat = esp.lonlat_h[column_idx * 2 + 1] * 180 / M_PI;

                    // get col mu star from zenith angle

                    std::shared_ptr<double[]> pressure_int_h    = pressure_int.get_host_data();
                    std::shared_ptr<double[]> temperature_int_h = temperature_int.get_host_data();


                    double p_toa = pressure_int_h[esp.nvi - 1];
                    double p_boa = pressure_int_h[0];


                    // Print out initial TP profile
                    string output_file_name = DBG_OUTPUT_DIR + "tpprofile_interface.dat";

                    FILE*  tp_output_file = fopen(output_file_name.c_str(), "w");
                    string comment        = "# Helios TP interface profile table at lat: ["
                                     + std::to_string(lon) + "] lon: [" + std::to_string(lat)
                                     + "] mustar: [" + std::to_string(mu_star) + "] P_BOA: ["
                                     + std::to_string(p_boa) + "] P_TOA: [" + std::to_string(p_toa)
                                     + "]\n";

                    fprintf(tp_output_file, comment.c_str());
                    fprintf(tp_output_file, "#\tT[K]\tP[bar]\n");


                    for (int i = 0; i < esp.nvi; i++) {
                        fprintf(tp_output_file,
                                "%#.6g\t%#.6g\n",
                                temperature_int_h[i],
                                pressure_int_h[i] / 1e5);
                    }

                    fclose(tp_output_file);
                }
#endif // DUMP_HELIOS_TP

                // initialise delta_col_mass
                // TODO: should this go inside alf?
                // printf("initialise_delta_colmass\n");
                if (iso) {
                    initialise_delta_colmass_iso<<<((num_layers + 1) / num_blocks) + 1,
                                                   num_blocks>>>(
                        *alf.delta_col_mass, *pressure_int, gravit, num_layers);
                }
                else {
                    initialise_delta_colmass_noniso<<<((num_layers + 1) / num_blocks) + 1,
                                                      num_blocks>>>(*alf.delta_col_upper,
                                                                    *alf.delta_col_lower,
                                                                    column_layer_pressure,
                                                                    *pressure_int,
                                                                    gravit,
                                                                    num_layers);
                }
                hipDeviceSynchronize();
                cuda_check_status_or_exit(__FILE__, __LINE__);
                // printf("initialise_delta_colmass done\n");

                // get z_lay
                // TODO: z_lay for beam computation
                // TODO: check how it is used and check that it doesn't interpolate to interface
                //        in which case we need to pass z_int
                double* z_lay = esp.Altitude_d;
                double* z_int = esp.Altitudeh_d;
                // internal to alfrodull_engine

                double* dev_starflux = *star_flux;
                // limit where to switch from noniso to iso equations to keep model stable
                // as defined in host_functions.set_up_numerical_parameters
                double delta_tau_limit = 1e-4;

                // compute fluxes

                // Check in here, some values from initial setup might change per column: e.g. mu_star;
                //printf("compute_radiative_transfer\n");


                // singlewalk
                //  true -> 201 iterations,
                //  false -> 4 iterations,

                bool    singlewalk_loc    = scat_single_walk;
                int     ninterface        = nlayer + 1;
                int     column_offset_int = column_idx * ninterface;
                double* F_col_down_tot    = &((*F_down_tot)[column_offset_int]);
                double* F_col_up_tot      = &((*F_up_tot)[column_offset_int]);
                double* F_col_dir_tot     = &((*F_dir_tot)[column_offset_int]);
                double* F_col_net         = &((*F_net)[column_offset_int]);

                //            double* F_dir_band_col    = &((*F_dir_band)[ninterface * nbin]);
                double* F_dir_band_col = &((*F_dir_band)[0]);

                double* F_up_TOA_spectrum_col = &((*F_up_TOA_spectrum)[column_idx * nbin]);

                alf.compute_radiative_transfer(dev_starflux,          // dev_starflux
                                               *temperature_lay,      // dev_T_lay
                                               *temperature_int,      // dev_T_int
                                               column_layer_pressure, // dev_p_lay
                                               *pressure_int,         // dev_p_int
                                               false,                 // interp_press_and_temp
                                               true,                  // interp_and_calc_flux_step
                                               z_lay,                 // z_lay
                                               singlewalk_loc,        // singlewalk
                                               *F_down_wg,
                                               *F_up_wg,
                                               *Fc_down_wg,
                                               *Fc_up_wg,
                                               *F_dir_wg,
                                               *Fc_dir_wg,
                                               delta_tau_limit,
                                               F_col_down_tot,
                                               F_col_up_tot,
                                               F_col_dir_tot,
                                               F_col_net,
                                               *F_down_band,
                                               *F_up_band,
                                               F_dir_band_col,
                                               F_up_TOA_spectrum_col,
                                               mu_star);
                hipDeviceSynchronize();
                cuda_check_status_or_exit(__FILE__, __LINE__);
                // get the g0 and w0 integrated
                if (store_w0_g0) {
                    // TODO could be optimised by storing band values and integrate only on output
                    // but takes up more space
                    double* g0_tot_col = &((*g0_tot)[column_idx * nlayer * nbin]);
                    double* w0_tot_col = &((*w0_tot)[column_idx * nlayer * nbin]);
                    alf.get_column_integrated_g0_w0(g0_tot_col, w0_tot_col);
                }
                // compute Delta flux

                // set Qheat
                //printf("increment_column_Qheat\n");
                double* qheat = &((*Qheat)[column_offset]);
                compute_column_Qheat<<<(esp.nv / num_blocks) + 1,
                                       num_blocks>>>(F_col_net, // net flux, layer
                                                     z_int,
                                                     qheat,
                                                     F_intern,
                                                     num_layers);
                hipDeviceSynchronize();
                cuda_check_status_or_exit(__FILE__, __LINE__);

#ifdef DEBUG_PRINTOUT_ARRAYS
                debug_print_columns(esp, -col_cos_zenith_angle_h[column_idx], nstep, column_idx);
#endif // DEBUG_PRINTOUT_ARRAYS
                hipDeviceSynchronize();
                cuda_check_status_or_exit(__FILE__, __LINE__);
            }
            start_up = false;
        }

        printf("\r\n");

        int num_samples = (esp.point_num * nlayer);
        increment_Qheat<<<(num_samples / num_blocks) + 1, num_blocks>>>(
            esp.profx_Qheat_d, *Qheat, qheat_scaling, num_samples);
        hipDeviceSynchronize();
        cuda_check_status_or_exit(__FILE__, __LINE__);
    }
    last_step = nstep;

    BENCH_POINT_I(nstep, "Alf_phy_loop_E", (), ("F_up_tot", "F_down_tot", "AlfQheat"));

    return true;
}

bool two_streams_radiative_transfer::store_init(storage& s) {
    if (!s.has_table("/Tstar"))
        s.append_value(T_star, "/Tstar", "K", "Temperature of host star");
    // s.append_value(Tint, "/Tint", "K", "Temperature of interior heat flux");
    if (!s.has_table("/planet_star_dist"))
        s.append_value(planet_star_dist_config,
                       "/planet_star_dist",
                       "au",
                       "distance b/w host star and planet");

    if (!s.has_table("/radius_star"))
        s.append_value(R_star_config, "/radius_star", "R_sun", "radius of host star");

    s.append_value(iso ? 1.0 : 0.0, "/alf_isothermal", "-", "Isothermal layers");
    s.append_value(
        real_star ? 1.0 : 0.0, "/alf_real_star", "-", "Alfrodull use real star spectrum or Planck");
    s.append_value(
        fake_opac ? 1.0 : 0.0, "/alf_fake_opac", "-", "Alfrodull use artificial opacity");
    s.append_value(scat ? 1.0 : 0.0, "/alf_scat", "-", "Scattering");
    s.append_value(
        scat_corr ? 1.0 : 0.0, "/alf_scat_corr", "-", "Improved two-stream scattering correction");

    s.append_value(g_0, "/alf_g_0", "-", "asymmetry factor");
    s.append_value(diffusivity, "/alf_diffusivity", "-", "Diffusivity factor");
    s.append_value(epsi, "/alf_epsi", "-", "One over Diffusivity factor");

    s.append_value(alf.opacities.ny, "/alf_ny", "-", "number of weights in bins");

    s.append_value(i2s_transition, "/alf_i2s_transition", "-", "i2s transition");

    s.append_value(compute_every_n_iteration,
                   "/alf_compute_periodicity",
                   "n",
                   "Alfrodull compute periodicity");
    //s.append_value(opacities_file, "/alf_opacity_file", "path", "Alfrodull opacitiy file used");


    s.append_value(dir_beam ? 1.0 : 0.0, "/alf_dir_beam", "-", "Direct irradiation beam");
    s.append_value(geom_zenith_corr ? 1.0 : 0.0,
                   "/alf_geom_zenith_corr",
                   "-",
                   "Geometric zenith angle correction");

    s.append_value(alf.opacities.nbin, "/alf_num_bands", "-", "Number of wavelength_bands for Alf");
    s.append_value(
        store_w0_g0 ? 1.0 : 0.0, "/alf_w0_g0_per_band", "-", "Stored w0 and g0 per band for Alf");


    s.append_value(clouds ? 1.0 : 0.0, "/alf_cloud", "-", "Simulate clouds");
    s.append_value(fcloud, "/alf_fcloud", "-", "f_cloud");

    return true;
}
//***************************************************************************************************

bool two_streams_radiative_transfer::store(const ESP& esp, storage& s) {
    std::shared_ptr<double[]> F_net_h = F_net.get_host_data();
    s.append_table(F_net_h.get(), F_net.get_size(), "/F_net", "W m^-2", "Net Flux");

    std::shared_ptr<double[]> Qheat_h = Qheat.get_host_data();
    s.append_table(Qheat_h.get(), Qheat.get_size(), "/Alf_Qheat", "W m^-3", "Alfrodull Qheat");

    std::shared_ptr<double[]> F_up_tot_h = F_up_tot.get_host_data();
    s.append_table(
        F_up_tot_h.get(), F_up_tot.get_size(), "/F_up_tot", "W m^-2", "Total upward flux");

    std::shared_ptr<double[]> F_down_tot_h = F_down_tot.get_host_data();
    s.append_table(
        F_down_tot_h.get(), F_down_tot.get_size(), "/F_down_tot", "W m^-2", "Total downward flux");

    std::shared_ptr<double[]> F_dir_tot_h = F_dir_tot.get_host_data();
    s.append_table(
        F_dir_tot_h.get(), F_dir_tot.get_size(), "/F_dir_tot", "W m^-2", "Total beam flux");

    if (store_w0_g0) {
        std::shared_ptr<double[]> w0_tot_h = w0_tot.get_host_data();
        s.append_table(w0_tot_h.get(),
                       w0_tot.get_size(),
                       "/w0_band",
                       " ",
                       "Single scattering albedo per band");

        std::shared_ptr<double[]> g0_tot_h = g0_tot.get_host_data();
        s.append_table(g0_tot_h.get(), g0_tot.get_size(), "/g0_band", " ", "asymmetry per band");
    }


    std::shared_ptr<double[]> F_up_TOA_spectrum_h = F_up_TOA_spectrum.get_host_data();
    s.append_table(F_up_TOA_spectrum_h.get(),
                   F_up_TOA_spectrum.get_size(),
                   "/F_up_TOA_spectrum",
                   "W m^-2",
                   "Upward Flux per bin at TOA");

    std::shared_ptr<double[]> lambda_wave_h = alf.opacities.dev_opac_wave.get_host_data();
    s.append_table(lambda_wave_h.get(),
                   alf.opacities.dev_opac_wave.get_size(),
                   "/lambda_wave",
                   "m",
                   "Center wavelength");

    std::shared_ptr<double[]> lambda_interwave_h = alf.opacities.dev_opac_interwave.get_host_data();
    s.append_table(lambda_interwave_h.get(),
                   alf.opacities.dev_opac_interwave.get_size(),
                   "/lambda_interwave",
                   "m",
                   "Interface wavelength");

    std::shared_ptr<double[]> lambda_deltawave_h = alf.opacities.dev_opac_deltawave.get_host_data();
    s.append_table(lambda_deltawave_h.get(),
                   alf.opacities.dev_opac_deltawave.get_size(),
                   "/lambda_deltawave",
                   "m",
                   "Wavelength width of bins");

    s.append_value(qheat_scaling, "/qheat_scaling", "-", "QHeat scaling");

    return true;
}


bool two_streams_radiative_transfer::free_memory() {

    return true;
}

// ***************************************************************************************************************
void two_streams_radiative_transfer::print_weighted_band_data_to_file(
    ESP&                        esp,
    int                         nstep,
    int                         column_idx,
    cuda_device_memory<double>& array,
    string                      output_file_base) {
    int nbin = alf.opacities.nbin;
    int ny   = alf.opacities.ny;

    std::string DBG_OUTPUT_DIR = esp.get_output_dir()
                                 + "/alfprof/"
                                   "step_"
                                 + std::to_string(nstep) + "/column_" + std::to_string(column_idx)
                                 + "/";
    create_output_dir(DBG_OUTPUT_DIR);

    // Print out single scattering albedo data

    int                       num_val = array.get_size() / (nbin * ny);
    std::shared_ptr<double[]> array_h =
        integrate_band(*array, *alf.gauss_weights, num_val, nbin, ny);


    string output_file_name = DBG_OUTPUT_DIR + output_file_base + ".dat";

    FILE* output_file = fopen(output_file_name.c_str(), "w");
    // std::shared_ptr<double[]> opac_wg_lay_h =
    //     get_cuda_data(*alf.opac_wg_lay, esp.nv * nbin);

    std::shared_ptr<double[]> delta_lambda_h =
        get_cuda_data(*alf.opacities.dev_opac_deltawave, nbin);

    fprintf(output_file, "bin\t");
    fprintf(output_file, "deltalambda\t");
    for (int i = 0; i < esp.nv; i++)
        fprintf(output_file, "layer[%d]\t", i);
    fprintf(output_file, "\n");

    for (int b = 0; b < nbin; b++) {
        fprintf(output_file, "%d\t", b);
        fprintf(output_file, "%#.6g\t", delta_lambda_h[b]);
        for (int i = 0; i < esp.nv; i++) {
            fprintf(output_file, "%#.6g\t", array_h[b + i * nbin]);
        }
        fprintf(output_file, "\n");
    }
    fclose(output_file);

    cuda_check_status_or_exit((string(__FILE__ ":") + string(output_file_base)).c_str(), __LINE__);
}

// ***************************************************************************************************************
// Helper function to print out all datasets for debugging and comparisong to HELIOS
void two_streams_radiative_transfer::debug_print_columns(ESP&   esp,
                                                         double cmustar,
                                                         int    nstep,
                                                         int    column_idx) {
    int nbin = alf.opacities.nbin;
    int ny   = alf.opacities.ny;

    std::string DBG_OUTPUT_DIR = esp.get_output_dir()
                                 + "/alfprof/"
                                   "step_"
                                 + std::to_string(nstep) + "/column_" + std::to_string(column_idx)
                                 + "/";
    create_output_dir(DBG_OUTPUT_DIR);

    {


        double lon = esp.lonlat_h[column_idx * 2 + 0] * 180 / M_PI;
        double lat = esp.lonlat_h[column_idx * 2 + 1] * 180 / M_PI;


        // Print out initial TP profile
        string output_file_name = DBG_OUTPUT_DIR + "tprofile_interp.dat";

        FILE*  tp_output_file = fopen(output_file_name.c_str(), "w");
        string comment = "# Helios TP profile table at lat: [" + std::to_string(lon) + "] lon: ["
                         + std::to_string(lat) + "] mustar: [" + std::to_string(cmustar) + "]\n";

        fprintf(tp_output_file, comment.c_str());
        fprintf(tp_output_file, "#\tT[K]\n");


        std::shared_ptr<double[]> temperature_h = get_cuda_data(*temperature_lay, esp.nv + 1);

        fprintf(tp_output_file, "BOA\t%#.6g\n", temperature_h[esp.nv]);
        for (int i = 0; i < esp.nv; i++) {
            fprintf(tp_output_file, "%d\t%#.6g\n", i, temperature_h[i]);
        }

        fclose(tp_output_file);
    }


    {

        // Print out planck data

        string output_file_name = DBG_OUTPUT_DIR + "plkprofile.dat";

        FILE*                     planck_output_file = fopen(output_file_name.c_str(), "w");
        std::shared_ptr<double[]> planck_h =
            get_cuda_data(*alf.planckband_lay, (esp.nv + 2) * nbin);

        std::shared_ptr<double[]> delta_lambda_h =
            get_cuda_data(*alf.opacities.dev_opac_deltawave, nbin);

        fprintf(planck_output_file, "bin\t");
        fprintf(planck_output_file, "deltalambda\t");
        for (int i = 0; i < esp.nv; i++)
            fprintf(planck_output_file, "layer[%d]\t", i);
        fprintf(planck_output_file, "layer[TOA]\t");
        fprintf(planck_output_file, "layer[BOA]\t");
        fprintf(planck_output_file, "\n");

        for (int b = 0; b < nbin; b++) {
            fprintf(planck_output_file, "%d\t", b);
            fprintf(planck_output_file, "%#.6g\t", delta_lambda_h[b]);
            for (int i = 0; i < esp.nv + 2; i++) {
                fprintf(planck_output_file, "%#.6g\t", planck_h[b * (esp.nv + 2) + i]);
            }
            fprintf(planck_output_file, "\n");
        }
        fclose(planck_output_file);
    }

    {
        // Print out mean molecular weight data

        string output_file_name        = DBG_OUTPUT_DIR + "meanmolmassprofile.dat";
        FILE*  meanmolmass_output_file = fopen(output_file_name.c_str(), "w");

        std::shared_ptr<double[]> meanmolmass_h = get_cuda_data(*alf.meanmolmass_lay, (esp.nv));

        fprintf(meanmolmass_output_file, "layer\t");
        fprintf(meanmolmass_output_file, "meanmolmass\n");

        for (int i = 0; i < esp.nv; i++) {
            fprintf(meanmolmass_output_file, "%d\t", i);
            fprintf(meanmolmass_output_file, "%#.6g\n", meanmolmass_h[i] / AMU);
        }

        fclose(meanmolmass_output_file);
    }

    {
        // Print out mean molecular weight data

        string output_file_name = DBG_OUTPUT_DIR + "deltacolmassprofile.dat";

        FILE*                     deltacolmass_output_file = fopen(output_file_name.c_str(), "w");
        std::shared_ptr<double[]> deltacolmass_h = get_cuda_data(*alf.delta_col_mass, (esp.nv));

        fprintf(deltacolmass_output_file, "layer\t");
        fprintf(deltacolmass_output_file, "delta_col_mass\n");

        for (int i = 0; i < esp.nv; i++) {
            fprintf(deltacolmass_output_file, "%d\t", i);
            fprintf(deltacolmass_output_file, "%#.6g\n", deltacolmass_h[i]);
        }

        fclose(deltacolmass_output_file);
    }

    {

        // Print out opacities data

        int                       num_val = alf.opac_wg_lay.get_size() / (nbin * ny);
        std::shared_ptr<double[]> opac_h =
            integrate_band(*alf.opac_wg_lay, *alf.gauss_weights, num_val, nbin, ny);


        string output_file_name = DBG_OUTPUT_DIR + "opacprofile.dat";

        FILE* opac_output_file = fopen(output_file_name.c_str(), "w");

        std::shared_ptr<double[]> delta_lambda_h =
            get_cuda_data(*alf.opacities.dev_opac_deltawave, nbin);

        fprintf(opac_output_file, "bin\t");
        fprintf(opac_output_file, "deltalambda\t");
        for (int i = 0; i < esp.nv; i++)
            fprintf(opac_output_file, "layer[%d]\t", i);
        fprintf(opac_output_file, "\n");

        for (int b = 0; b < nbin; b++) {
            fprintf(opac_output_file, "%d\t", b);
            fprintf(opac_output_file, "%#.6g\t", delta_lambda_h[b]);
            for (int i = 0; i < esp.nv; i++) {
                fprintf(opac_output_file, "%#.6g\t", opac_h[b + i * nbin]);
            }
            fprintf(opac_output_file, "\n");
        }
        fclose(opac_output_file);
    }

    if (iso) {
        // Print out optical depth data
        int                       num_val = alf.delta_tau_wg.get_size() / (nbin * ny);
        std::shared_ptr<double[]> delta_tau_h =
            integrate_band(*alf.delta_tau_wg, *alf.gauss_weights, num_val, nbin, ny);


        string output_file_name = DBG_OUTPUT_DIR + "opt_depthprofile.dat";

        FILE* opt_depth_output_file = fopen(output_file_name.c_str(), "w");
        // std::shared_ptr<double[]> opac_wg_lay_h =
        //     get_cuda_data(*alf.opac_wg_lay, esp.nv * nbin);

        std::shared_ptr<double[]> delta_lambda_h =
            get_cuda_data(*alf.opacities.dev_opac_deltawave, nbin);

        fprintf(opt_depth_output_file, "bin\t");
        fprintf(opt_depth_output_file, "deltalambda\t");
        for (int i = 0; i < esp.nv; i++)
            fprintf(opt_depth_output_file, "layer[%d]\t", i);
        fprintf(opt_depth_output_file, "\n");

        for (int b = 0; b < nbin; b++) {
            fprintf(opt_depth_output_file, "%d\t", b);
            fprintf(opt_depth_output_file, "%#.6g\t", delta_lambda_h[b]);
            for (int i = 0; i < esp.nv; i++) {
                fprintf(opt_depth_output_file, "%#.6g\t", delta_tau_h[b + i * nbin]);
            }
            fprintf(opt_depth_output_file, "\n");
        }
        fclose(opt_depth_output_file);
    }
    else {
        {
            // Print out optical depth data
            int                       num_val = alf.delta_tau_wg_upper.get_size() / (nbin * ny);
            std::shared_ptr<double[]> delta_tau_h =
                integrate_band(*alf.delta_tau_wg_upper, *alf.gauss_weights, num_val, nbin, ny);


            string output_file_name = DBG_OUTPUT_DIR + "opt_depth_upper_profile.dat";

            FILE* opt_depth_output_file = fopen(output_file_name.c_str(), "w");
            // std::shared_ptr<double[]> opac_wg_lay_h =
            //     get_cuda_data(*alf.opac_wg_lay, esp.nv * nbin);

            std::shared_ptr<double[]> delta_lambda_h =
                get_cuda_data(*alf.opacities.dev_opac_deltawave, nbin);

            fprintf(opt_depth_output_file, "bin\t");
            fprintf(opt_depth_output_file, "deltalambda\t");
            for (int i = 0; i < esp.nv; i++)
                fprintf(opt_depth_output_file, "layer[%d]\t", i);
            fprintf(opt_depth_output_file, "\n");

            for (int b = 0; b < nbin; b++) {
                fprintf(opt_depth_output_file, "%d\t", b);
                fprintf(opt_depth_output_file, "%#.6g\t", delta_lambda_h[b]);
                for (int i = 0; i < esp.nv; i++) {
                    fprintf(opt_depth_output_file, "%#.6g\t", delta_tau_h[b + i * nbin]);
                }
                fprintf(opt_depth_output_file, "\n");
            }
            fclose(opt_depth_output_file);
        }
        {
            // Print out optical depth data
            int                       num_val = alf.delta_tau_wg_lower.get_size() / (nbin * ny);
            std::shared_ptr<double[]> delta_tau_h =
                integrate_band(*alf.delta_tau_wg_lower, *alf.gauss_weights, num_val, nbin, ny);


            string output_file_name = DBG_OUTPUT_DIR + "opt_depth_lower_profile.dat";

            FILE* opt_depth_output_file = fopen(output_file_name.c_str(), "w");
            // std::shared_ptr<double[]> opac_wg_lay_h =
            //     get_cuda_data(*alf.opac_wg_lay, esp.nv * nbin);

            std::shared_ptr<double[]> delta_lambda_h =
                get_cuda_data(*alf.opacities.dev_opac_deltawave, nbin);

            fprintf(opt_depth_output_file, "bin\t");
            fprintf(opt_depth_output_file, "deltalambda\t");
            for (int i = 0; i < esp.nv; i++)
                fprintf(opt_depth_output_file, "layer[%d]\t", i);
            fprintf(opt_depth_output_file, "\n");

            for (int b = 0; b < nbin; b++) {
                fprintf(opt_depth_output_file, "%d\t", b);
                fprintf(opt_depth_output_file, "%#.6g\t", delta_lambda_h[b]);
                for (int i = 0; i < esp.nv; i++) {
                    fprintf(opt_depth_output_file, "%#.6g\t", delta_tau_h[b + i * nbin]);
                }
                fprintf(opt_depth_output_file, "\n");
            }
            fclose(opt_depth_output_file);
        }
    }

    if (iso) {
        // Print out transmission data

        int                       num_val = alf.trans_wg.get_size() / (nbin * ny);
        std::shared_ptr<double[]> trans_h =
            integrate_band(*alf.trans_wg, *alf.gauss_weights, num_val, nbin, ny);


        string output_file_name = DBG_OUTPUT_DIR + "trans_band_profile.dat";

        FILE* trans_band_output_file = fopen(output_file_name.c_str(), "w");
        // std::shared_ptr<double[]> opac_wg_lay_h =
        //     get_cuda_data(*alf.opac_wg_lay, esp.nv * nbin);

        std::shared_ptr<double[]> delta_lambda_h =
            get_cuda_data(*alf.opacities.dev_opac_deltawave, nbin);

        fprintf(trans_band_output_file, "bin\t");
        fprintf(trans_band_output_file, "deltalambda\t");
        for (int i = 0; i < esp.nv; i++)
            fprintf(trans_band_output_file, "layer[%d]\t", i);
        fprintf(trans_band_output_file, "\n");

        for (int b = 0; b < nbin; b++) {
            fprintf(trans_band_output_file, "%d\t", b);
            fprintf(trans_band_output_file, "%#.6g\t", delta_lambda_h[b]);
            for (int i = 0; i < esp.nv; i++) {
                fprintf(trans_band_output_file, "%#.6g\t", trans_h[b + i * nbin]);
            }
            fprintf(trans_band_output_file, "\n");
        }
        fclose(trans_band_output_file);
    }
    else {
        {
            // Print out transmission data

            int                       num_val = alf.trans_wg_upper.get_size() / (nbin * ny);
            std::shared_ptr<double[]> trans_h =
                integrate_band(*alf.trans_wg_upper, *alf.gauss_weights, num_val, nbin, ny);


            string output_file_name = DBG_OUTPUT_DIR + "trans_band_upper_profile.dat";

            FILE* trans_band_output_file = fopen(output_file_name.c_str(), "w");
            // std::shared_ptr<double[]> opac_wg_lay_h =
            //     get_cuda_data(*alf.opac_wg_lay, esp.nv * nbin);

            std::shared_ptr<double[]> delta_lambda_h =
                get_cuda_data(*alf.opacities.dev_opac_deltawave, nbin);

            fprintf(trans_band_output_file, "bin\t");
            fprintf(trans_band_output_file, "deltalambda\t");
            for (int i = 0; i < esp.nv; i++)
                fprintf(trans_band_output_file, "layer[%d]\t", i);
            fprintf(trans_band_output_file, "\n");

            for (int b = 0; b < nbin; b++) {
                fprintf(trans_band_output_file, "%d\t", b);
                fprintf(trans_band_output_file, "%#.6g\t", delta_lambda_h[b]);
                for (int i = 0; i < esp.nv; i++) {
                    fprintf(trans_band_output_file, "%#.6g\t", trans_h[b + i * nbin]);
                }
                fprintf(trans_band_output_file, "\n");
            }
            fclose(trans_band_output_file);
        }

        {
            // Print out transmission data

            int                       num_val = alf.trans_wg_lower.get_size() / (nbin * ny);
            std::shared_ptr<double[]> trans_h =
                integrate_band(*alf.trans_wg_lower, *alf.gauss_weights, num_val, nbin, ny);


            string output_file_name = DBG_OUTPUT_DIR + "trans_band_lower_profile.dat";

            FILE* trans_band_output_file = fopen(output_file_name.c_str(), "w");
            // std::shared_ptr<double[]> opac_wg_lay_h =
            //     get_cuda_data(*alf.opac_wg_lay, esp.nv * nbin);

            std::shared_ptr<double[]> delta_lambda_h =
                get_cuda_data(*alf.opacities.dev_opac_deltawave, nbin);

            fprintf(trans_band_output_file, "bin\t");
            fprintf(trans_band_output_file, "deltalambda\t");
            for (int i = 0; i < esp.nv; i++)
                fprintf(trans_band_output_file, "layer[%d]\t", i);
            fprintf(trans_band_output_file, "\n");

            for (int b = 0; b < nbin; b++) {
                fprintf(trans_band_output_file, "%d\t", b);
                fprintf(trans_band_output_file, "%#.6g\t", delta_lambda_h[b]);
                for (int i = 0; i < esp.nv; i++) {
                    fprintf(trans_band_output_file, "%#.6g\t", trans_h[b + i * nbin]);
                }
                fprintf(trans_band_output_file, "\n");
            }
            fclose(trans_band_output_file);
        }
    }

    if (iso) {
        // Print out single scattering albedo data

        int                       num_val = alf.w0_wg.get_size() / (nbin * ny);
        std::shared_ptr<double[]> w0_h =
            integrate_band(*alf.w0_wg, *alf.gauss_weights, num_val, nbin, ny);


        string output_file_name = DBG_OUTPUT_DIR + "single_scat_band_profile.dat";

        FILE* singscat_output_file = fopen(output_file_name.c_str(), "w");
        // std::shared_ptr<double[]> opac_wg_lay_h =
        //     get_cuda_data(*alf.opac_wg_lay, esp.nv * nbin);

        std::shared_ptr<double[]> delta_lambda_h =
            get_cuda_data(*alf.opacities.dev_opac_deltawave, nbin);

        fprintf(singscat_output_file, "bin\t");
        fprintf(singscat_output_file, "deltalambda\t");
        for (int i = 0; i < esp.nv; i++)
            fprintf(singscat_output_file, "layer[%d]\t", i);
        fprintf(singscat_output_file, "\n");

        for (int b = 0; b < nbin; b++) {
            fprintf(singscat_output_file, "%d\t", b);
            fprintf(singscat_output_file, "%#.6g\t", delta_lambda_h[b]);
            for (int i = 0; i < esp.nv; i++) {
                fprintf(singscat_output_file, "%#.6g\t", w0_h[b + i * nbin]);
            }
            fprintf(singscat_output_file, "\n");
        }
        fclose(singscat_output_file);
    }
    else {
        {
            // Print out single scattering albedo data

            int                       num_val = alf.w0_wg_upper.get_size() / (nbin * ny);
            std::shared_ptr<double[]> w0_h =
                integrate_band(*alf.w0_wg_upper, *alf.gauss_weights, num_val, nbin, ny);


            string output_file_name = DBG_OUTPUT_DIR + "single_scat_band_upper_profile.dat";

            FILE* singscat_output_file = fopen(output_file_name.c_str(), "w");
            // std::shared_ptr<double[]> opac_wg_lay_h =
            //     get_cuda_data(*alf.opac_wg_lay, esp.nv * nbin);

            std::shared_ptr<double[]> delta_lambda_h =
                get_cuda_data(*alf.opacities.dev_opac_deltawave, nbin);

            fprintf(singscat_output_file, "bin\t");
            fprintf(singscat_output_file, "deltalambda\t");
            for (int i = 0; i < esp.nv; i++)
                fprintf(singscat_output_file, "layer[%d]\t", i);
            fprintf(singscat_output_file, "\n");

            for (int b = 0; b < nbin; b++) {
                fprintf(singscat_output_file, "%d\t", b);
                fprintf(singscat_output_file, "%#.6g\t", delta_lambda_h[b]);
                for (int i = 0; i < esp.nv; i++) {
                    fprintf(singscat_output_file, "%#.6g\t", w0_h[b + i * nbin]);
                }
                fprintf(singscat_output_file, "\n");
            }
            fclose(singscat_output_file);
        }

        {
            // Print out single scattering albedo data

            int                       num_val = alf.w0_wg_lower.get_size() / (nbin * ny);
            std::shared_ptr<double[]> w0_h =
                integrate_band(*alf.w0_wg_lower, *alf.gauss_weights, num_val, nbin, ny);


            string output_file_name = DBG_OUTPUT_DIR + "single_scat_band_lower_profile.dat";

            FILE* singscat_output_file = fopen(output_file_name.c_str(), "w");
            // std::shared_ptr<double[]> opac_wg_lay_h =
            //     get_cuda_data(*alf.opac_wg_lay, esp.nv * nbin);

            std::shared_ptr<double[]> delta_lambda_h =
                get_cuda_data(*alf.opacities.dev_opac_deltawave, nbin);

            fprintf(singscat_output_file, "bin\t");
            fprintf(singscat_output_file, "deltalambda\t");
            for (int i = 0; i < esp.nv; i++)
                fprintf(singscat_output_file, "layer[%d]\t", i);
            fprintf(singscat_output_file, "\n");

            for (int b = 0; b < nbin; b++) {
                fprintf(singscat_output_file, "%d\t", b);
                fprintf(singscat_output_file, "%#.6g\t", delta_lambda_h[b]);
                for (int i = 0; i < esp.nv; i++) {
                    fprintf(singscat_output_file, "%#.6g\t", w0_h[b + i * nbin]);
                }
                fprintf(singscat_output_file, "\n");
            }
            fclose(singscat_output_file);
        }
    }
    //***********************************************************************************************
    if (iso) {
        print_weighted_band_data_to_file(esp, nstep, column_idx, alf.M_term, "M_profile");
        print_weighted_band_data_to_file(esp, nstep, column_idx, alf.N_term, "N_profile");
        print_weighted_band_data_to_file(esp, nstep, column_idx, alf.P_term, "P_profile");
        print_weighted_band_data_to_file(esp, nstep, column_idx, alf.G_plus, "G_plus_profile");
        print_weighted_band_data_to_file(esp, nstep, column_idx, alf.G_minus, "G_minus_profile");
    }
    else {
        print_weighted_band_data_to_file(esp, nstep, column_idx, alf.M_upper, "M_upper_profile");
        print_weighted_band_data_to_file(esp, nstep, column_idx, alf.M_lower, "M_lower_profile");

        print_weighted_band_data_to_file(esp, nstep, column_idx, alf.N_upper, "N_upper_profile");
        print_weighted_band_data_to_file(esp, nstep, column_idx, alf.N_lower, "N_lower_profile");

        print_weighted_band_data_to_file(esp, nstep, column_idx, alf.P_upper, "P_upper_profile");
        print_weighted_band_data_to_file(esp, nstep, column_idx, alf.P_lower, "P_lower_profile");

        print_weighted_band_data_to_file(
            esp, nstep, column_idx, alf.G_plus_upper, "G_plus_upper_profile");
        print_weighted_band_data_to_file(
            esp, nstep, column_idx, alf.G_plus_lower, "G_plus_lower_profile");

        print_weighted_band_data_to_file(
            esp, nstep, column_idx, alf.G_minus_upper, "G_minus_upper_profile");
        print_weighted_band_data_to_file(
            esp, nstep, column_idx, alf.G_minus_lower, "G_minus_lower_profile");
    }
    //***********************************************************************************************
    {
        // Print out downward flux

        int                       num_val = F_down_wg.get_size() / (nbin * ny);
        std::shared_ptr<double[]> fd_h =
            integrate_band(*F_down_wg, *alf.gauss_weights, num_val, nbin, ny);


        string output_file_name = DBG_OUTPUT_DIR + "F_down_profile.dat";

        FILE* Fd_output_file = fopen(output_file_name.c_str(), "w");
        // std::shared_ptr<double[]> opac_wg_lay_h =
        //     get_cuda_data(*alf.opac_wg_lay, esp.nv * nbin);

        std::shared_ptr<double[]> delta_lambda_h =
            get_cuda_data(*alf.opacities.dev_opac_deltawave, nbin);

        fprintf(Fd_output_file, "bin\t");
        fprintf(Fd_output_file, "deltalambda\t");
        for (int i = 0; i < esp.nvi; i++)
            fprintf(Fd_output_file, "interface[%d]\t", i);
        fprintf(Fd_output_file, "\n");

        for (int b = 0; b < nbin; b++) {
            fprintf(Fd_output_file, "%d\t", b);
            fprintf(Fd_output_file, "%#.6g\t", delta_lambda_h[b]);
            for (int i = 0; i < esp.nvi; i++) {
                fprintf(Fd_output_file, "%#.6g\t", fd_h[b + i * nbin]);
            }
            fprintf(Fd_output_file, "\n");
        }
        fclose(Fd_output_file);
    }

    {
        // Print out downward flux

        int                       num_val = F_up_wg.get_size() / (nbin * ny);
        std::shared_ptr<double[]> fu_h =
            integrate_band(*F_up_wg, *alf.gauss_weights, num_val, nbin, ny);


        string output_file_name = DBG_OUTPUT_DIR + "F_up_profile.dat";

        FILE* Fu_output_file = fopen(output_file_name.c_str(), "w");
        // std::shared_ptr<double[]> opac_wg_lay_h =
        //     get_cuda_data(*alf.opac_wg_lay, esp.nv * nbin);

        std::shared_ptr<double[]> delta_lambda_h =
            get_cuda_data(*alf.opacities.dev_opac_deltawave, nbin);

        fprintf(Fu_output_file, "bin\t");
        fprintf(Fu_output_file, "deltalambda\t");
        for (int i = 0; i < esp.nvi; i++)
            fprintf(Fu_output_file, "interface[%d]\t", i);
        fprintf(Fu_output_file, "\n");

        for (int b = 0; b < nbin; b++) {
            fprintf(Fu_output_file, "%d\t", b);
            fprintf(Fu_output_file, "%#.6g\t", delta_lambda_h[b]);
            for (int i = 0; i < esp.nvi; i++) {
                fprintf(Fu_output_file, "%#.6g\t", fu_h[b + i * nbin]);
            }
            fprintf(Fu_output_file, "\n");
        }
        fclose(Fu_output_file);
    }

    {
        // Print out direct beam flux

        int                       num_val = F_dir_wg.get_size() / (nbin * ny);
        std::shared_ptr<double[]> fdir_h =
            integrate_band(*F_dir_wg, *alf.gauss_weights, num_val, nbin, ny);


        string output_file_name = DBG_OUTPUT_DIR + "F_dir_profile.dat";

        FILE* Fdir_output_file = fopen(output_file_name.c_str(), "w");
        // std::shared_ptr<double[]> opac_wg_lay_h =
        //     get_cuda_data(*alf.opac_wg_lay, esp.nv * nbin);

        std::shared_ptr<double[]> delta_lambda_h =
            get_cuda_data(*alf.opacities.dev_opac_deltawave, nbin);

        fprintf(Fdir_output_file, "bin\t");
        fprintf(Fdir_output_file, "deltalambda\t");
        for (int i = 0; i < esp.nvi; i++)
            fprintf(Fdir_output_file, "interface[%d]\t", i);
        fprintf(Fdir_output_file, "\n");

        for (int b = 0; b < nbin; b++) {
            fprintf(Fdir_output_file, "%d\t", b);
            fprintf(Fdir_output_file, "%#.6g\t", delta_lambda_h[b]);
            for (int i = 0; i < esp.nvi; i++) {
                fprintf(Fdir_output_file, "%#.6g\t", fdir_h[b + i * nbin]);
            }
            fprintf(Fdir_output_file, "\n");
        }
        fclose(Fdir_output_file);
    }


    {
        // Print out alf qheat

        int num_val = F_dir_wg.get_size() / (nbin * ny);


        int col_offset = column_idx * esp.nv;

        string output_file_name = DBG_OUTPUT_DIR + "alf_qheat_profile.dat";

        FILE* output_file = fopen(output_file_name.c_str(), "w");

        std::shared_ptr<double[]> qh_h = get_cuda_data(&((Qheat.ptr())[col_offset]), esp.nv);

        fprintf(output_file, "level\tqheat\n");

        for (int i = 0; i < esp.nv; i++) {
            fprintf(output_file, "%d\t%#.6g\n", i, qh_h[i]);
        }

        fclose(output_file);
    }
}
